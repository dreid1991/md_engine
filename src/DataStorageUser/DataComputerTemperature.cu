#include "hip/hip_runtime.h"
#include "DataComputerTemperature.h"
#include "cutils_func.h"
#include "boost_for_export.h"
#include "State.h"
namespace py = boost::python;
using namespace MD_ENGINE;

DataComputerTemperature::DataComputerTemperature(State *state_, bool computeScalar_, bool computeTensor_) : DataComputer(state_, computeScalar_, computeTensor_, false, false) {
}


void DataComputerTemperature::computeScalar_GPU(bool transferToCPU, uint32_t groupTag) {
    GPUData &gpd = state->gpd;
    tempGPUScalar.d_data.memset(0);
    lastGroupTag = groupTag;
    int nAtoms = state->atoms.size();
    if (groupTag == 1) {
         accumulate_gpu<float, float4, SumVectorSqr3DOverW, N_DATA_PER_THREAD> <<<NBLOCK(nAtoms / (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*PERBLOCK*sizeof(float)>>>
            (tempGPUScalar.getDevData(), state->gpd.vs.getDevData(), nAtoms, state->devManager.prop.warpSize, SumVectorSqr3DOverW());
    } else {
        accumulate_gpu_if<float, float4, SumVectorSqr3DOverWIf, N_DATA_PER_THREAD> <<<NBLOCK(nAtoms / (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*PERBLOCK*sizeof(float)>>>
            (tempGPUScalar.getDevData(), gpd.vs.getDevData(), nAtoms, state->devManager.prop.warpSize, SumVectorSqr3DOverWIf(gpd.fs.getDevData(), groupTag));
    }
    if (transferToCPU) {
        //does NOT sync
        tempGPUScalar.dataToHost();
    }
}



void DataComputerTemperature::computeTensor_GPU(bool transferToCPU, uint32_t groupTag) {
    GPUData &gpd = state->gpd;
    tempGPUTensor.d_data.memset(0); 
    lastGroupTag = groupTag;
    int nAtoms = state->atoms.size();
    if (groupTag == 1) {
        accumulate_gpu<Virial, float4, SumVectorToVirial, N_DATA_PER_THREAD>  <<<NBLOCK(nAtoms / (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*PERBLOCK*sizeof(Virial)>>>
            (tempGPUTensor.getDevData(), gpd.vs.getDevData(), nAtoms, state->devManager.prop.warpSize, SumVectorToVirial());    
    } else {
        accumulate_gpu_if<Virial, float4, SumVectorToVirialIf, N_DATA_PER_THREAD> <<<NBLOCK(nAtoms / (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*PERBLOCK*sizeof(Virial)>>>
            (tempGPUTensor.getDevData(), gpd.vs.getDevData(), nAtoms, state->devManager.prop.warpSize, SumVectorToVirialIf(gpd.fs.getDevData(), groupTag));
    } 
    if (transferToCPU) {
        //does NOT sync
        tempGPUTensor.dataToHost();
    }
}

void DataComputerTemperature::computeScalar_CPU() {
    int n;
    double total = tempGPUScalar.h_data[0];
    if (lastGroupTag == 1) {
        n = state->atoms.size();//* (int *) &tempGPUScalar.h_data[1];
    } else {
        n = * (int *) &tempGPUScalar.h_data[1];
    }
    if (state->is2d) {
        ndf = 2*n;
    } else {
        ndf = 3*n;
    }
    totalKEScalar = total;
    tempScalar = total / ndf; 
}

void DataComputerTemperature::computeTensor_CPU() {
    int n;
    Virial total = tempGPUTensor.h_data[0];
    if (lastGroupTag == 1) {
        n = state->atoms.size();
    } else {
        n = * (int *) &tempGPUTensor.h_data[1];
    }
    total /= n;
    tempTensor = total;
}

void DataComputerTemperature::appendScalar(boost::python::list &vals) {
    vals.append(tempScalar);
}
void DataComputerTemperature::appendTensor(boost::python::list &vals) {
    vals.append(tempTensor);
}

void DataComputerTemperature::prepareForRun() {
    if (computingScalar) {
        tempGPUScalar = GPUArrayGlobal<float>(2);
    }
    if (computingTensor) {
        tempGPUTensor = GPUArrayGlobal<Virial>(2);
    }
}

