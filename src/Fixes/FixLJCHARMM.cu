#include "hip/hip_runtime.h"
#include "FixLJCHARMM.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "State.h"
#include "cutils_func.h"
#include "ReadConfig.h"
#include "EvaluatorWrapper.h"
#include "PairEvaluatorCHARMM.h"
#include "EvaluatorWrapper.h"
//#include "ChargeEvaluatorEwald.h"
namespace py = boost::python;
const std::string LJCHARMMType = "LJCHARMM";



FixLJCHARMM::FixLJCHARMM(boost::shared_ptr<State> state_, std::string handle_, std::string mixingRules_)
    : FixPair(state_, handle_, "all", LJCHARMMType, true, false, 1, mixingRules_),
    epsHandle("eps"), sigHandle("sig"), eps14Handle("eps14"), sig14Handle("sig14"), rCutHandle("rCut")
{

    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(eps14Handle, epsilons14);
    initializeParameters(sig14Handle, sigmas14);
    initializeParameters(rCutHandle, rCuts);
    paramOrder = {rCutHandle, epsHandle, sigHandle, eps14Handle, sig14Handle};
    readFromRestart();
    canAcceptChargePairCalc = true;
    setEvalWrapper();
}

    //neighbor coefs are not used in CHARMM force field, because it specifies 1-4 sigmas and epsilons.
    //These parameters will be ignored in the evaluator
    // but we need to tell the evaluator if it's a 1-4 neighbor.  We do this by making a dummy neighborCoefs array, where all the values are 1 except the 1-4 value, which is zero.
void FixLJCHARMM::compute(int virialMode) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    auto neighborCoefs = state->specialNeighborCoefs;
    evalWrap->compute(nAtoms,nPerRingPoly, gpd.xs(activeIdx), gpd.fs(activeIdx),
                      neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
                      state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU,
                      neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.virials.d_data.data(), gpd.qs(activeIdx), chargeRCut, virialMode, nThreadPerBlock(), nThreadPerAtom());

}

void FixLJCHARMM::singlePointEng(real *perParticleEng) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    auto neighborCoefs = state->specialNeighborCoefs;
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    //real neighborCoefs[4] = {1, 1, 1, 0}; //see comment above
    //evalWrap->energy(nAtoms,nPerRingPoly, gpd.xs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut);
    evalWrap->energy(nAtoms,nPerRingPoly, gpd.xs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut, nThreadPerBlock(), nThreadPerAtom());
}


void FixLJCHARMM::singlePointEngGroupGroup(real *perParticleEng, uint32_t tagA, uint32_t tagB) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    auto neighborCoefs = state->specialNeighborCoefs;
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    //real neighborCoefs[4] = {1, 1, 1, 0}; //see comment above
    //evalWrap->energy(nAtoms,nPerRingPoly, gpd.xs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut);
    evalWrap->energyGroupGroup(nAtoms,nPerRingPoly, gpd.xs(activeIdx), gpd.fs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut, tagA, tagB, nThreadPerBlock(), nThreadPerAtom());
}

void FixLJCHARMM::setEvalWrapper() {
    if (evalWrapperMode == "offload") {
        EvaluatorCHARMM eval(state->specialNeighborCoefs[2]);
        // <EVALUATOR, N_PARAM, COMP_PAIRS> ... should make an assert for future dev that n_param == 5, in case it changes
        evalWrap = pickEvaluator<EvaluatorCHARMM, 5, true>(eval, chargeCalcFix);
    } else if (evalWrapperMode == "self") {
        EvaluatorCHARMM eval(state->specialNeighborCoefs[2]);
        evalWrap = pickEvaluator<EvaluatorCHARMM, 5, true>(eval, nullptr);
    }
}

bool FixLJCHARMM::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillGeo = [] (real a, real b) {
        return sqrt(a*b);
    };

    auto fillArith = [] (real a, real b) {
        return (a+b) / 2.0;
    };
    auto fillRCut = [this] (real a, real b) {
        return (real) std::fmax(a, b);
    };
    auto none = [] (real a){};

    auto fillRCutDiag = [this] () {
        return (real) state->rCut;
    };

    auto processEps = [] (real a) {
        return 24*a;
    };
    auto processSig = [] (real a) {
        return pow(a, 6);
    };
    auto processRCut = [] (real a) {
        return a*a;
    };

    auto copyEpsDiag = [&] () {
    };
    //copy in non 1-4 parameters for sig, eps

    std::vector<real> &epsPreProc = *paramMap["eps"];
    std::vector<real> &eps14PreProc = *paramMap["eps14"];

    std::vector<real> &sigPreProc = *paramMap["sig"];
    std::vector<real> &sig14PreProc = *paramMap["sig14"];
    assert(epsPreProc.size() == sigPreProc.size());
    int numTypes = state->atomParams.numTypes;
    for (int i=0; i<state->atomParams.numTypes; i++) {

        if (squareVectorRef<real>(eps14PreProc.data(), numTypes, i, i) == DEFAULT_FILL) {
            squareVectorRef<real>(eps14PreProc.data(), numTypes, i, i) = squareVectorRef<real>(epsPreProc.data(), numTypes, i, i) * state->specialNeighborCoefs[2]; 
        }

        if (squareVectorRef<real>(sig14PreProc.data(), numTypes, i, i) == DEFAULT_FILL) {
            squareVectorRef<real>(sig14PreProc.data(), numTypes, i, i) = squareVectorRef<real>(sigPreProc.data(), numTypes, i, i) * state->specialNeighborCoefs[2]; 
        }
    }


    prepareParameters(epsHandle, fillGeo, processEps, false);
    prepareParameters(eps14Handle, fillGeo, processEps, false);

	if (mixingRules==ARITHMETICTYPE) {
		prepareParameters(sigHandle, fillArith, processSig, false);
		prepareParameters(sig14Handle, fillArith, processSig, false);
	} else {
		prepareParameters(sigHandle, fillGeo, processSig, false);
		prepareParameters(sig14Handle, fillGeo, processSig, false);
	}
    prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);

    sendAllToDevice();
    setEvalWrapper();
    for (int i=0; i<2; i++) {
        if (state->specialNeighborCoefs[i] == state->specialNeighborCoefs[2]) {
            printf("Warning: FixLJCHARMM complains that 1-%d special neighbor coef is the same as the 1-4 coefficient.  Your 1-%d interactions will use 1-4 coefficients.\n", i+1, i+1);
        }
    }
    if (state->specialNeighborCoefs[2] == 0) {
        printf("Warning: FixLJCharmm complains that 1-4 neighbor coefficients cannot be 0\n");
        assert(state->specialNeighborCoefs[2] != 0);
    }
    prepared = true;
    return prepared;
}

std::string FixLJCHARMM::restartChunk(std::string format) {
    std::stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}



void FixLJCHARMM::addSpecies(std::string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(eps14Handle, epsilons14);
    initializeParameters(sig14Handle, sigmas14);
    initializeParameters(rCutHandle, rCuts);

}

std::vector<real> FixLJCHARMM::getRCuts() { 
    std::vector<real> res;
    std::vector<real> &src = *(paramMap[rCutHandle]);
    for (real x : src) {
        if (x == DEFAULT_FILL) {
            res.push_back(-1);
        } else {
            res.push_back(x);
        }
    }

    return res;
}

void export_FixLJCHARMM() {
    py::class_<FixLJCHARMM, boost::shared_ptr<FixLJCHARMM>, py::bases<FixPair>, boost::noncopyable > (
        "FixLJCHARMM",
        py::init<boost::shared_ptr<State>, std::string, py::optional<std::string> > (py::args("state", "handle", "mixingRules"))
    )
      ;

}
