#include "hip/hip_runtime.h"
#include "FixWCA.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "PairEvaluateIso.h"
#include "State.h"
#include "cutils_func.h"
#include "EvaluatorWrapper.h"

const std::string LJCutType = "LJCutWCA";
namespace py = boost::python;

FixWCA::FixWCA(SHARED(State) state_, std::string handle_, std::string mixingRules_)
    : FixPair(state_, handle_, "all", LJCutType, true, false, 1, ARITHMETICTYPE),
      epsHandle("eps"), sigHandle("sig"), rCutHandle("rCut") {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    paramOrder = {rCutHandle, epsHandle, sigHandle};
    readFromRestart();
    setEvalWrapper();
}
void FixWCA::compute(int virialMode) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    real *neighborCoefs = state->specialNeighborCoefs;


    evalWrap->compute(nAtoms,nPerRingPoly, gpd.xs(activeIdx), gpd.fs(activeIdx),
                      neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
                      state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU,
                      neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.virials.d_data.data(), gpd.qs(activeIdx), chargeRCut, virialMode, nThreadPerBlock(), nThreadPerAtom());



}

void FixWCA::singlePointEng(real *perParticleEng) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    real *neighborCoefs = state->specialNeighborCoefs;

    evalWrap->energy(nAtoms,nPerRingPoly, gpd.xs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut, nThreadPerBlock(), nThreadPerAtom());



}

bool FixWCA::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillGeo = [] (real a, real b) {
        return sqrt(a*b);
    };

    auto fillArith = [] (real a, real b) {
        return (a+b) / 2.0;
    };
//     auto fillRCut = [this] (real a, real b) {
//         return (real) std::fmax(a, b);
//     };
    auto none = [] (real a){};

    auto fillRCutDiag = [this] () {
        return (real) state->rCut;
    };

    auto processEps = [] (real a) {
        return 24*a;
    };
    auto processSig = [] (real a) {
        return pow(a, 6);
    };
    auto processRCut = [] (real a) {
        return a*a;
    };

    auto fillRCut = [this] (int a, int b) {
        int numTypes = state->atomParams.numTypes;
        real sig = squareVectorRef<real>(paramMap[sigHandle]->data(),numTypes,a,b);
        return sig*pow(2.0,1.0/6.0);
    };    
    prepareParameters(epsHandle, fillGeo, processEps, false);
	if (mixingRules=="arithmetic") {
		prepareParameters(sigHandle, fillArith, processSig, false);
	} else {
		prepareParameters(sigHandle, fillGeo, processSig, false);
	}
    prepareParameters_from_other(rCutHandle, fillRCut, processRCut, false);

//     prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);
    sendAllToDevice();
    setEvalWrapper();
    prepared = true;
    return prepared;
}

void FixWCA::setEvalWrapper() {
    if (evalWrapperMode == "offload") {
        EvaluatorWCA eval;
        evalWrap = pickEvaluator<EvaluatorWCA, 3, true>(eval, chargeCalcFix);
    } else if (evalWrapperMode == "self") {
        EvaluatorWCA eval;
        evalWrap = pickEvaluator<EvaluatorWCA, 3, true>(eval, nullptr);
    }

}

std::string FixWCA::restartChunk(std::string format) {
    std::stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}



void FixWCA::addSpecies(std::string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);

}

std::vector<real> FixWCA::getRCuts() {
    std::vector<real> res;
    std::vector<real> &src = *(paramMap[rCutHandle]);
    for (real x : src) {
        if (x == DEFAULT_FILL) {
            res.push_back(-1);
        } else {
            res.push_back(x);
        }
    }

    return res;
}

bool FixWCA::setParameter(std::string param,
                           std::string handleA,
                           std::string handleB,
                           double val)
{
      if (param==sigHandle) FixPair::setParameter(rCutHandle, handleA,handleB,val*pow(2.0,1.0/6.0));
      return FixPair::setParameter(param, handleA,handleB,val);
      
}

void export_FixWCA() {
    py::class_<FixWCA,
                          SHARED(FixWCA),
                          py::bases<FixPair>, boost::noncopyable > (
        "FixWCA",
        py::init<SHARED(State), std::string, py::optional<std::string> > (
            py::args("state", "handle", "mixingRules")))
        .def("setParameter", &FixWCA::setParameter,
                ( py::arg("param"),
                  py::arg("handleA"),
                  py::arg("handleB"),
                  py::arg("val"))
            )
        ;

}

/*
void export_FixWCA() {
    py::class_<FixWCA,
                          SHARED(FixWCA),
                          py::bases<FixPair>, boost::noncopyable > (
        "FixWCA",
        py::init<SHARED(State), std::string, py::optional<std::string> > (
            py::args("state", "handle", "mixingRules")))
        .def("setParameter", &FixWCA::setParameter,
                ( py::arg("param"),
                  py::arg("handleA"),
                  py::arg("handleB"),
                  py::arg("val"))
            )
        ;

}
*/
