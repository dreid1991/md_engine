#include "hip/hip_runtime.h"
#include "FixNVTAndersen.h"

#define INVALID_VAL INT_MAX
#include "Bounds.h"
#include "cutils_func.h"
#include "State.h"

namespace py=boost::python;


void FixNVTAndersen::setDefaults() {
    seed=0;
}

const std::string NVTAndersenType = "NVTAndersen";

FixNVTAndersen::FixNVTAndersen(SHARED(State) state_, std::string handle_, std::string groupHandle_, py::list intervals_, py::list temps_, real nu_, int applyEvery_)
    : Fix(state_, handle_, groupHandle_, NVTAndersenType, false, false, false, applyEvery_),
      Interpolator(intervals_, temps_), 
      tempComputer(state, "scalar")
{
    setDefaults();
    isThermostat = true;
    nudt         = state_->dt * nu_; 
}

FixNVTAndersen::FixNVTAndersen(SHARED(State) state_, std::string handle_, std::string groupHandle_, py::object tempFunc_, real nu_, int applyEvery_)
    : Fix(state_, handle_, groupHandle_, NVTAndersenType, false, false, false, applyEvery_),
      Interpolator(tempFunc_), 
      tempComputer(state, "scalar")
{
    setDefaults();
    isThermostat = true;
    nudt         = state_->dt * nu_; 
}

FixNVTAndersen::FixNVTAndersen(SHARED(State) state_, std::string handle_, std::string groupHandle_, double constTemp_, real nu_, int applyEvery_)
    : Fix(state_, handle_, groupHandle_, NVTAndersenType, false, false, false, applyEvery_),
      Interpolator(constTemp_), 
      tempComputer(state, "scalar")
{
    setDefaults();
    isThermostat = true;
    nudt         = state_->dt * nu_; 
}

void __global__ initRand(int nAtoms, hiprandState_t *states, int seed,int turn) {
    int idx = GETIDX();
    hiprand_init(seed, idx, turn, states + idx);

}

bool FixNVTAndersen::prepareForRun() {
    turnBeginRun = state->runInit;
    turnFinishRun = state->runInit + state->runningFor;
    tempComputer.prepareForRun();
    randStates = GPUArrayDeviceGlobal<hiprandState_t>(state->atoms.size());
    initRand<<<NBLOCK(state->atoms.size()), PERBLOCK>>>(state->atoms.size(), randStates.data(), seed,state->turn);
    prepared = true;
    return prepared;
}

void __global__ resample_no_tags_cu(int nAtoms, real4 *vs, hiprandState_t *randStates, real tempSet, real nudt, real boltz, real mvv_to_e) {
    int idx = GETIDX();
    if (tempSet > 0 and idx < nAtoms) {
        hiprandState_t *randState = randStates + idx;
        hiprandState_t localState=*randState;
        if ( hiprand_uniform(&localState) <= nudt ) {
            // resample from Boltzmann distribution
            real4 vnew    = vs[idx];
            real  invmass = vnew.w;
            real  sigma   = sqrtf(boltz * tempSet * invmass / mvv_to_e);
                real sx; real sy; real sz;
                sx = hiprand_normal(&localState);
                sy = hiprand_normal(&localState);
                sz = hiprand_normal(&localState);
                vnew.x = sigma*sx;
                vnew.y = sigma*sy;
                vnew.z = sigma*sz;
            vs[idx]= vnew;
        }
        *randState=localState;
    }
}

void __global__ resample_cu(int nAtoms, uint groupTag, real4 *vs, real4 *fs, hiprandState_t *randStates, real tempSet, real nudt, real boltz, real mvv_to_e) {

    int idx = GETIDX();
    if (tempSet > 0 and idx < nAtoms) {
        hiprandState_t *randState = randStates + idx;
        hiprandState_t localState=*randState;
        uint groupTagAtom = ((uint *) (fs+idx))[3];
        if (groupTag & groupTagAtom) {
            if ( hiprand_uniform(&localState) <= nudt ) {
                // resample from Boltzmann distribution
                real4 vnew    = vs[idx];
                real  invmass = vnew.w;
                real  sigma   = sqrtf(boltz * tempSet * invmass / mvv_to_e);
                real sx; real sy; real sz;
                sx = hiprand_normal(&localState);
                sy = hiprand_normal(&localState);
                sz = hiprand_normal(&localState);
                vnew.x = sigma*sx;
                vnew.y = sigma*sy;
                vnew.z = sigma*sz;
                vs[idx]= vnew;
            }
            *randState=localState;
        }
    }
}

void FixNVTAndersen::compute(int virialMode) {

    tempComputer.computeScalar_GPU(true, groupTag);
    int nAtoms    = state->atoms.size();
    int64_t turn  = state->turn;
    computeCurrentVal(turn);
    double temp   = getCurrentVal();
    GPUData &gpd  = state->gpd;
    int activeIdx = gpd.activeIdx();

    if (groupTag == 1) {
        resample_no_tags_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.vs(activeIdx), randStates.data(), 
                temp, nudt,state->units.boltz,state->units.mvv_to_eng);

    } else {
        resample_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, groupTag, gpd.vs(activeIdx),gpd.fs(activeIdx), randStates.data(), 
                temp, nudt,state->units.boltz,state->units.mvv_to_eng);
    }
}


bool FixNVTAndersen::postRun() {
    finishRun();
    prepared = false;
    return true;
}

void FixNVTAndersen::setParams(double seed_) {
    if (seed_ != INVALID_VAL) {
        seed = seed_;
    }
}

Interpolator *FixNVTAndersen::getInterpolator(std::string type) {
    if (type == "temp") {
        return (Interpolator *) this;
    }
    return nullptr;
}


void export_FixNVTAndersen() {
    py::class_<FixNVTAndersen, SHARED(FixNVTAndersen), py::bases<Fix>, boost::noncopyable > (
        "FixNVTAndersen", 
        py::init<boost::shared_ptr<State>, std::string, std::string, py::list, py::list, py::optional<real,int> >(
            py::args("state", "handle", "groupHandle", "intervals", "temps","nu", "applyEvery")
            )

        
    )
   
    .def(py::init<boost::shared_ptr<State>, std::string, std::string, py::object, py::optional<real,int> >(
                
            py::args("state", "handle", "groupHandle", "tempFunc","nu","applyEvery")
                )
            )
    .def(py::init<boost::shared_ptr<State>, std::string, std::string, double, py::optional<real,int> >(
            py::args("state", "handle", "groupHandle", "temp","nu", "applyEvery")
                )
            )
    .def("setParameters", &FixNVTAndersen::setParams,
         (py::arg("seed") = INVALID_VAL)
        )
    ;
}
