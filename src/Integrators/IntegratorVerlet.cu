#include "hip/hip_runtime.h"
#include "IntegratorVerlet.h"

#include <chrono>

#include <boost/python.hpp>
#include <boost/shared_ptr.hpp>

#include "Logging.h"
#include "State.h"

namespace py = boost::python;

__global__ void preForce_cu(int nAtoms, float4 *xs, float4 *vs, float4 *fs,
                            float dt)
{
    int idx = GETIDX();
    if (idx < nAtoms) {
        // Update velocity by a half timestep
        float4 vel = vs[idx];
        float invmass = vel.w;

        float4 force = fs[idx];

        float3 dv = 0.5f * dt * invmass * make_float3(force);
        vel += dv;
        vs[idx] = vel;

        // Update position by a full timestep
        float4 pos = xs[idx];

        float3 dx = dt*make_float3(vel);
        pos += dx;
        xs[idx] = pos;

        // Set forces to zero before force calculation
        fs[idx] = make_float4(0.0f, 0.0f, 0.0f, force.w);
    }
}

__global__ void postForce_cu(int nAtoms, float4 *vs, float4 *fs, float dt)
{
    int idx = GETIDX();
    if (idx < nAtoms) {
        // Update velocities by a halftimestep
        float4 vel = vs[idx];
        float invmass = vel.w;

        float4 force = fs[idx];

        float3 dv = 0.5f * dt * invmass * make_float3(force);
        vel += dv;
        vs[idx] = vel;
    }
}

IntegratorVerlet::IntegratorVerlet(State *state_)
    : Integrator(state_)
{

}
void IntegratorVerlet::run(int numTurns)
{

    basicPreRunChecks();
    basicPrepare(numTurns);

    int periodicInterval = state->periodicInterval;

    auto start = std::chrono::high_resolution_clock::now();
    bool computeVirialsInForce = state->dataManager.computeVirialsInForce;
    for (int i=0; i<numTurns; ++i) {
        if (state->turn % periodicInterval == 0) {
            state->gridGPU.periodicBoundaryConditions();
        }
        // Prepare for timestep
        //! \todo Should asyncOperations() and doDataCollection() go into
        //!       Integrator::stepInit()? Same for periodicBoundayConditions()

        stepInit(computeVirialsInForce);

        // Perform first half of velocity-Verlet step
        preForce();

        // Recalculate forces
        force(computeVirialsInForce);
        asyncOperations();
        doDataComputation();

        // Perform second half of velocity-Verlet step
        postForce();

        stepFinal();
        doDataAppending();

        //! \todo The following parts could also be moved into stepFinal
        state->turn++;
        if (state->verbose && (i+1 == numTurns || state->turn % state->shoutEvery == 0)) {
            mdMessage("Turn %d %.2f percent done.\n", (int)state->turn, 100.0*(i+1)/numTurns);
        }
    }

    //! \todo These parts could be moved to basicFinish()
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    mdMessage("runtime %f\n%e particle timesteps per second\n",
              duration.count(), state->atoms.size()*numTurns / duration.count());

    basicFinish();
}

void IntegratorVerlet::preForce()
{
    uint activeIdx = state->gpd.activeIdx();
    preForce_cu<<<NBLOCK(state->atoms.size()), PERBLOCK>>>(
            state->atoms.size(),
            state->gpd.xs.getDevData(),
            state->gpd.vs.getDevData(),
            state->gpd.fs.getDevData(),
            state->dt);
}

void IntegratorVerlet::postForce()
{
    uint activeIdx = state->gpd.activeIdx();
    postForce_cu<<<NBLOCK(state->atoms.size()), PERBLOCK>>>(
            state->atoms.size(),
            state->gpd.vs.getDevData(),
            state->gpd.fs.getDevData(),
            state->dt);
}

void export_IntegratorVerlet()
{
    py::class_<IntegratorVerlet,
               boost::shared_ptr<IntegratorVerlet>,
               py::bases<Integrator>,
               boost::noncopyable>
    (
        "IntegratorVerlet",
        py::init<State *>()
    )
    .def("run", &IntegratorVerlet::run)
    ;
}
